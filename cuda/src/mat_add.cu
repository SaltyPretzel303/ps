#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include "cuda_help.h"

#define BLOCK_DIM 32

__global__ void add_matrices(int *mat_a, int *mat_b, int *mat_c, int *avs, int mat_dim)
{

	__shared__ int sh_res[BLOCK_DIM];

	int gl_ind = blockDim.x * blockIdx.x + threadIdx.x;

	if (gl_ind < mat_dim * mat_dim)
	{

		int value = mat_a[gl_ind] + mat_b[gl_ind];
		mat_c[gl_ind] = value;

		sh_res[threadIdx.x] = gl_ind;
	}

	__syncthreads();

	int my_row = gl_ind / mat_dim;

	

	return;
}

int main(void)
{

	int mat_dim = 100;

	int *mat_a = (int *)malloc(mat_dim * mat_dim * sizeof(int));
	int *mat_b = (int *)malloc(mat_dim * mat_dim * sizeof(int));
	int *mat_c = (int *)malloc(mat_dim * mat_dim * sizeof(int));
	int *row_avs = (int *)malloc(mat_dim * sizeof(int));

	init_vec(mat_a, mat_dim * mat_dim, 1);
	init_vec(mat_b, mat_dim * mat_dim, 1);
	init_vec(mat_c, mat_dim * mat_dim, 0);
	init_vec(row_avs, mat_dim, 0);

	int *dev_mat_a;
	cuda_err(hipMalloc((void **)&dev_mat_a, mat_dim * mat_dim * sizeof(int)),
			 "mat_a hipMalloc");

	int *dev_mat_b;
	cuda_err(hipMalloc((void **)&dev_mat_b, mat_dim * mat_dim * sizeof(int)),
			 "mat_b hipMalloc");

	int *dev_mat_c;
	cuda_err(hipMalloc((void **)&dev_mat_c, mat_dim * mat_dim * sizeof(int)),
			 "mat_c hipMalloc");

	int *dev_row_avs;
	cuda_err(hipMalloc((void **)&dev_row_avs, mat_dim * sizeof(int)),
			 "dev_row_aws hipMalloc");

	cuda_err(hipMemcpy(dev_mat_a, mat_a, mat_dim * mat_dim * sizeof(int), hipMemcpyHostToDevice),
			 "mat_a hipMemcpy");

	cuda_err(hipMemcpy(dev_mat_b, mat_b, mat_dim * mat_dim * sizeof(int), hipMemcpyHostToDevice),
			 "mat_b hipMemcpy");

	int grid_dim = (mat_dim * mat_dim + BLOCK_DIM) / BLOCK_DIM;

	add_matrices<<<grid_dim, BLOCK_DIM>>>(dev_mat_a, dev_mat_b, dev_mat_c, dev_row_avs, mat_dim);

	cuda_err(hipMemcpy(mat_c, dev_mat_c, mat_dim * mat_dim * sizeof(int), hipMemcpyDeviceToHost),
			 "mat_c hipMemcpy");
	cuda_err(hipMemcpy(row_avs, dev_row_avs, mat_dim * sizeof(int), hipMemcpyDeviceToHost),
			 "row_avs hipMemcpy");

	free(mat_a);
	free(mat_b);

	hipFree(dev_mat_a);
	hipFree(dev_mat_b);
	hipFree(dev_mat_c);
	hipFree(dev_row_avs);

	hipDeviceReset();

	for (int i = 0; i < mat_dim * mat_dim; i++)
	{
		if (mat_c[i] != 2)
		{
			printf("Failed mat at: [%d][%d] req: %d got: %d \n",
				   i / mat_dim, i % mat_dim, 2, mat_c[i]);

			free(mat_c);
			return 1;
		}
	}
	free(mat_c);

	printf("Matrix success ... \n");

	for (int i = 0; i < mat_dim; i++)
	{
		if (row_avs[i] != 1)
		{
			printf("Failed avs at: %d ... \n", i);

			free(row_avs);
			return 1;
		}
	}
	free(row_avs);
	printf("Averages success ... \n");

	return 0;
}