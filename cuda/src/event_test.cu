#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

__global__ void kernel(hipEvent_t s_event)
{
	return;
}

int main(void)
{

	hipEvent_t s_event;

	if (hipEventQuery(s_event) == hipSuccess)
	{
		printf("Before record success ... \n");
	}
	else
	{
		printf("Before record failed ... \n");
	}

	// hipEventRecord(s_event);

	kernel<<<1, 1>>>(s_event);

	hipEventCreate(&s_event);

	hipEventRecord()

		if (hipEventQuery(s_event) == hipSuccess)
	{
		printf("After record success ... \n");
	}
	else
	{
		printf("After record failed ... \n");
	}

	return 0;
}
